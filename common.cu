#include <string.h>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <stdbool.h>

#include "common.h"
#include "cudaErr.h"
#include "const.h"

void init_com (int **com)
{
  printf ("init communicator\n");
  int *com_t;
  gpuErrchk (hipHostMalloc (&com_t, sizeof(int) * (NB_BLOCKS + 1)));  // last byte to notify PK to exit

  for (int i = 0; i < NB_BLOCKS; i++)
    com_t[i] = 0;

  *com = com_t;
}

void startGPU (int *com)
{
  // printf ("startGPU\n");
  // memset (com, 1, NB_BLOCKS);
  for (int i=0; i < NB_BLOCKS;i++)
    com[i] = 1;
}

void waitGPU (int *com)
{
  // printf ("waitGPU\n");
  int sum;
  do
  {
    sum = 0;
    asm volatile ("" ::: "memory");
    for (int i = 0; i < NB_BLOCKS; i++)
      sum |= com[i];
  }while (sum != 0);
}

void endGPU (int *com)
{
  printf ("cpu is ending GPU\n");
  com [NB_BLOCKS] = 1;
}
