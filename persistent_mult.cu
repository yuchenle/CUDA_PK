#include "hip/hip_runtime.h"
  #include "persistent_mult.cuh"
  #include <stdio.h>

  __device__ void waitCPU (int *com)
  {
    int block_id = blockIdx.x;
    // printf ("waitCPU by PK_mult, from block %d\n", block_id);

    while (com[block_id] == 0 && com[NB_BLOCKS] != 1)
      __threadfence();
  }

  __device__ void work_complete (int *com)
  {
    int block_id = blockIdx.x;
    // printf ("work complete from PK_mult\n");

    com [block_id] = 0;
  }

  __global__ void persistent_mult (GPU_DATA_TYPE *data, GPU_DATA_TYPE value, int *com)
  {
    int local_id = threadIdx.x;
    int iter = 0;
    while (com[NB_BLOCKS] != 1)
    {
      iter++;
      if (local_id == 0)
        waitCPU (com);

      __syncthreads();

      // cancelling point
      if (com [NB_BLOCKS] == 1)
        return;

      int global_id = blockDim.x * blockIdx.x + threadIdx.x;

      // for now, just vecDec
      for (; global_id < GPU_SIZE && global_id > 0; global_id += blockDim.x * gridDim.x)
      {
        data[global_id] -= value;
      }
     
      if (local_id == 0)
        // inform next CUDA kernel
        work_complete (com);
    }
  }
