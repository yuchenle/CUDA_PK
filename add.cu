#include <stdio.h>
#include <stdlib.h>
#include <sys/ipc.h>
#include <sys/shm.h>
#include <unistd.h>
#include <sys/types.h>
#include <errno.h>

#include "cudaErr.h"
#include "const.h"
#include "persistent_add.cuh"
#include "common.h"

// global variables to be initialized
int shmid;
TYPE *ptr;

void init ()
{
  // FILE to key
  key_t key = ftok (FILENAME, FILEID);
  if (key == -1) 
  {
    printf ("ftok failed, errno = %s\n", strerror(errno));
    exit (-1);
  }

  // getting SHM id
  printf (" size of shm is %zu\n", sizeof(TYPE));
  shmid = shmget (key, sizeof (TYPE), 0666|IPC_CREAT);
  if (shmid == -1) 
  {
    printf ("shmget failed, errno = %s\n", strerror(errno));
    exit (-1);
  }

  // attach the SHM to this process
  ptr = (TYPE *) shmat (shmid, (void *)0, 0);
}

void notifyNext ()
{
  // printf ("notifyNext by add\n");
  ptr->ready[SIZE-1] = 0;
}

void waitPrevStage ()
{
  while (ptr->ready[SIZE-1] != 1)
  {
    asm volatile ("":::"memory");
  }
}

// void init_com (int **com)
// {
//   printf ("init communicator\n");
// 
//   int *com_t;
//   gpuErrchk (hipHostMalloc (&com_t, sizeof(int) * (NB_BLOCKS + 1))); //last element to stop GPU
// 
//   for (int i = 0; i < NB_BLOCKS; i++)
//     com_t[i] = 0;
// 
//   *com = com_t;
// }

// void startGPU (int *com)
// {
//   // printf ("startGPU\n");
//   for (int i=0; i < NB_BLOCKS;i++)
//     com[i] = 1;
// }

// void waitGPU (int *com)
// {
//   // printf ("waitGPU\n");
//   int sum;
//   do  
//   {
//     sum = 0;
//     asm volatile ("" ::: "memory");
//     for (int i = 0; i < NB_BLOCKS; i++)
//       sum |= com[i];
//   }while (sum != 0);
// }
 
// void endGPU (int *com)
// {
//   printf ("cpu is ending GPU\n");
//   com [NB_BLOCKS] = 1;
// }

int main()
{
  // allocating shared memory, inter-process communication
  init ();

  // establish intra-process GPU & CPU communication method
  int *com;
  init_com (&com);

  // setting GPU data pointer
  gpuErrchk (hipHostRegister ((void *)ptr->data, sizeof (GPU_DATA_TYPE) * GPU_SIZE, hipHostRegisterMapped|hipHostRegisterPortable));
  gpuErrchk (hipHostGetDevicePointer (&(ptr->d_data), (void *)ptr->data, 0));

  // launching PK
  persistent_add<<<NB_BLOCKS, NB_TH>>> (ptr->d_data, 1, com);

  for (int i=0; i<NUM_ITER; i++)
  {
    // printf ("add begins one iter, data[GPU_SIZE-1] = %.2f\n", ptr->data[GPU_SIZE-1]);

    waitPrevStage ();

    startGPU (com);

    /* generate/receive new data for next iteration? */

    waitGPU (com);
    /* use of the result, print it for instance */
    notifyNext();
    // printf ("add ends one iter, data[GPU_SIZE-1] = %.2f\n", ptr->data[GPU_SIZE-1]);
  }
  endGPU (com);
  gpuErrchk (hipHostUnregister (ptr->data));
  gpuErrchk (hipHostFree (com));
  return 0;
}
