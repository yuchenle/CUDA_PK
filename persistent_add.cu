#include "hip/hip_runtime.h"
  #include "persistent_add.cuh"
  #include <stdio.h>

  __device__ void waitCPU (int *com)
  {
    int block_id = blockIdx.x;
    // printf ("waitCPU by PK_add\n");

    while (com[block_id] != 1 && com [NB_BLOCKS] != 1)
    {
      __threadfence();
      // printf ("waitMult by PK_add, data[0] = %d, block %d\n", data[0], block_id);
    }
  }

  __device__ void work_complete (int *com)
  {
    int block_id = blockIdx.x;
    //  printf ("work_complete by PK_add\n");

    com[block_id] = 0;
  }

  __global__ void persistent_add (GPU_DATA_TYPE *data, GPU_DATA_TYPE value, int *com)
  {
    int local_id = threadIdx.x;
    while (com[NB_BLOCKS] != 1)
    {
      if (local_id == 0)
        waitCPU (com);

      __syncthreads();

      // cancelling point
      if (com[NB_BLOCKS]==1)
        return;

      int global_id = blockDim.x * blockIdx.x + threadIdx.x;

      // for now, just vecAdd
      for (; global_id < GPU_SIZE && global_id > 0; global_id += blockDim.x * gridDim.x)
        data[global_id] += value;
      
      if (local_id == 0)
        work_complete (com);
    }
  }
