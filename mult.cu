#include <stdio.h>
#include <stdlib.h>
#include <sys/ipc.h>
#include <sys/shm.h>
#include <unistd.h>
#include <sys/types.h>
#include <errno.h>

#include "cudaErr.h"
#include "const.h"
#include "persistent_mult.cuh"
#include "common.h"

// global variables to be initialized
int shmid;
TYPE *ptr;
int *d_com;

void init_shm (TYPE *ptr)
{
  for (int i = 0; i < SIZE; i++)
    ptr->ready[i] = 0;
}

void init ()
{
  // FILE to key
  key_t key = ftok (FILENAME, FILEID);
  if (key == -1) 
  {
    printf ("ftok failed, errno = %s\n", strerror(errno));
    exit (-1);
  }

  // getting SHM id
  printf (" size of shm is %zu\n", sizeof(TYPE));
  shmid = shmget (key, sizeof (TYPE), 0666|IPC_CREAT);
  if (shmid == -1) 
  {
    printf ("shmget failed, errno = %s\n", strerror(errno));
    exit (-1);
  }

  // attach the SHM to this process
  ptr = (TYPE *) shmat (shmid, (void *)0, 0);

  init_shm (ptr);
}

void notifyNext ()
{
  // printf ("notifyNext by mult\n");
  ptr->ready[SIZE-1] = 1;
}

void waitPrevStage ()
{
  while (ptr->ready[SIZE-1] != 0)
  {
    asm volatile ("":::"memory");
  }
}

// void init_com (int **com)
// {
//   printf ("init communicator\n");
// 
//   int *com_t;
//   gpuErrchk (hipHostMalloc (&com_t, sizeof(int) * (NB_BLOCKS + 1))); //last element to stop GPU
// 
//   for (int i = 0; i < NB_BLOCKS; i++)
//     com_t[i] = 0;
// 
//   *com = com_t;
// }
 
// void startGPU (int *com)
// {
//   // printf ("startGPU\n");
//   for (int i=0; i < NB_BLOCKS;i++)
//     com[i] = 1;
// }
 
// void waitGPU (int *com)
// {
//   // printf ("waitGPU\n");
//   int sum;
//   do  
//   {
//     sum = 0;
//     for (int i = 0; i < NB_BLOCKS; i++)
//       sum |= com[i];
//   }while (sum != 0);
// }
 
// void endGPU (int *com)
// {
//   printf ("cpu is ending GPU\n");
//   com [NB_BLOCKS] = 1;
// }

int main()
{
  // allocating shared memory, inter-process communication
  init ();

  // establish intra-process GPU & CPU communication method
  int *com;
  init_com (&com);

  // setting GPU data pointer
  gpuErrchk (hipHostRegister ((void *)ptr->data, sizeof (GPU_DATA_TYPE) * GPU_SIZE, hipHostRegisterMapped|hipHostRegisterPortable));
  gpuErrchk (hipHostGetDevicePointer (&(ptr->d_data), (void *)ptr->data, 0));

  // launching PK
  persistent_mult<<<NB_BLOCKS, NB_TH>>> (ptr->d_data, 1, com);

  for (int i=0; i<NUM_ITER; i++)
  {
    struct timespec stamp, prev_stamp;
    clock_gettime (CLOCK_REALTIME, &prev_stamp);
    double wtime;

    // printf ("mult begins one iter, data[GPU_SIZE-1] = %.2f\n", ptr->data[GPU_SIZE-1]);

    waitPrevStage ();

    startGPU (com);

    /* generate/receive new data for next iteration? */

    waitGPU (com);
    /* use of the result, print it for instance */
    notifyNext();
    clock_gettime (CLOCK_REALTIME, &stamp);
    wtime = (stamp.tv_sec - prev_stamp.tv_sec) * 1000000 + (stamp.tv_nsec - prev_stamp.tv_nsec) / 1000;
    printf ("%.4f\n", wtime);

  }
  endGPU (com);
  printf ("after all, last element is %.4f\n", ptr->data[GPU_SIZE-1]);
  gpuErrchk (hipHostUnregister (ptr->data));
  gpuErrchk (hipHostFree (com));
  return 0;
}
